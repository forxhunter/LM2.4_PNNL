/*
 * University of Illinois Open Source License
 * Copyright 2008-2018 Luthey-Schulten Group,
 * All rights reserved.
 * 
 * Developed by: Luthey-Schulten Group
 * 			     University of Illinois at Urbana-Champaign
 * 			     http://www.scs.uiuc.edu/~schulten
 * 
 * Permission is hereby granted, free of charge, to any person obtaining a copy of
 * this software and associated documentation files (the Software), to deal with 
 * the Software without restriction, including without limitation the rights to 
 * use, copy, modify, merge, publish, distribute, sublicense, and/or sell copies 
 * of the Software, and to permit persons to whom the Software is furnished to 
 * do so, subject to the following conditions:
 * 
 * - Redistributions of source code must retain the above copyright notice, 
 * this list of conditions and the following disclaimers.
 * 
 * - Redistributions in binary form must reproduce the above copyright notice, 
 * this list of conditions and the following disclaimers in the documentation 
 * and/or other materials provided with the distribution.
 * 
 * - Neither the names of the Luthey-Schulten Group, University of Illinois at
 * Urbana-Champaign, nor the names of its contributors may be used to endorse or
 * promote products derived from this Software without specific prior written
 * permission.
 * 
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR 
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY, 
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT.  IN NO EVENT SHALL 
 * THE CONTRIBUTORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR 
 * OTHER LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, 
 * ARISING FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR 
 * OTHER DEALINGS WITH THE SOFTWARE.
 *
 * Author(s): Elijah Roberts
 */

#include "config.h"
#include "core/Types.h"
#include "core/Exceptions.h"
#include "rdme/CudaIntLattice.h"
#include "rdme/Lattice.h"

namespace lm {
namespace rdme {

CudaIntLattice::CudaIntLattice(lattice_coord_t size, si_dist_t latticeSpacing, uint particlesPerSite)
:IntLattice(size,latticeSpacing,particlesPerSite),cudaParticlesCurrent(0),cudaParticlesSize(0),cudaSiteTypesSize(0),cudaSiteTypes(NULL),isGPUMemorySynched(false)
{
    // Initialize the pointers.
    cudaParticles[0] = NULL;
    cudaParticles[1] = NULL;

    // Make sure the lattice dimensions are divisible by 32.
    if (size.x%32 != 0 || size.y%32 != 0 || size.z%32 != 0) throw InvalidArgException("size","each dimension of a CUDA lattice must be divisible by 32");
    allocateCudaMemory();
}

CudaIntLattice::CudaIntLattice(lattice_size_t xSize, lattice_size_t ySize, lattice_size_t zSize, si_dist_t latticeSpacing, uint particlesPerSite)
:IntLattice(xSize,ySize,zSize,latticeSpacing,particlesPerSite),cudaParticlesCurrent(0),cudaParticlesSize(0),cudaSiteTypesSize(0),cudaSiteTypes(NULL),isGPUMemorySynched(false)
{
    // Initialize the pointers.
    cudaParticles[0] = NULL;
    cudaParticles[1] = NULL;

    // Make sure the lattice dimensions are divisible by 32.
    if (size.x%32 != 0 || size.y%32 != 0 || size.z%32 != 0) throw InvalidArgException("size","each dimension of a CUDA lattice must be divisible by 32");
    allocateCudaMemory();
}

CudaIntLattice::~CudaIntLattice()
{
    deallocateCudaMemory();
}

void CudaIntLattice::allocateCudaMemory()
{
    // Allocate memory on the CUDA device.
    cudaParticlesSize=numberSites*wordsPerSite*sizeof(uint32_t);
    CUDA_EXCEPTION_CHECK(hipMalloc(&cudaParticles[0], cudaParticlesSize)); //TODO: track memory usage.
    CUDA_EXCEPTION_CHECK(hipMalloc(&cudaParticles[1], cudaParticlesSize)); //TODO: track memory usage.
    cudaSiteTypesSize=numberSites*sizeof(uint8_t);
    CUDA_EXCEPTION_CHECK(hipMalloc(&cudaSiteTypes, cudaSiteTypesSize)); //TODO: track memory usage.
}

void CudaIntLattice::deallocateCudaMemory()
{
    // If we have any allocated device memory, free it.
    if (cudaParticles[0] != NULL)
    {
        CUDA_EXCEPTION_CHECK(hipFree(cudaParticles[0])); //TODO: track memory usage.
        cudaParticles[0] = NULL;
    }
    if (cudaParticles[1] != NULL)
    {
        CUDA_EXCEPTION_CHECK(hipFree(cudaParticles[1])); //TODO: track memory usage.
        cudaParticles[1] = NULL;
    }
    cudaParticlesSize = 0;
    if (cudaSiteTypes != NULL)
    {
        CUDA_EXCEPTION_CHECK(hipFree(cudaSiteTypes)); //TODO: track memory usage.
        cudaSiteTypes = NULL;
        cudaSiteTypesSize = 0;
    }
}

void CudaIntLattice::copyToGPU()
{
	if (!isGPUMemorySynched)
	{
		CUDA_EXCEPTION_CHECK(hipMemcpy(cudaParticles[cudaParticlesCurrent], particles, cudaParticlesSize, hipMemcpyHostToDevice));
        CUDA_EXCEPTION_CHECK(hipMemcpy(cudaSiteTypes, siteTypes, cudaSiteTypesSize, hipMemcpyHostToDevice));
		isGPUMemorySynched = true;
	}
}

void CudaIntLattice::copyFromGPU()
{
	CUDA_EXCEPTION_CHECK(hipMemcpy(particles, cudaParticles[cudaParticlesCurrent], cudaParticlesSize, hipMemcpyDeviceToHost));
    CUDA_EXCEPTION_CHECK(hipMemcpy(siteTypes, cudaSiteTypes, cudaSiteTypesSize, hipMemcpyDeviceToHost));
	isGPUMemorySynched = true;
}

void * CudaIntLattice::getGPUMemorySrc()
{
    return cudaParticles[cudaParticlesCurrent];
}

void * CudaIntLattice::getGPUMemoryDest()
{
    return cudaParticles[cudaParticlesCurrent==0?1:0];
}

void CudaIntLattice::swapSrcDest()
{
    cudaParticlesCurrent = cudaParticlesCurrent==0?1:0;
}

void * CudaIntLattice::getGPUMemorySiteTypes()
{
    return cudaSiteTypes;
}

void CudaIntLattice::setSiteType(lattice_size_t x, lattice_size_t y, lattice_size_t z, site_t site) 
{
    IntLattice::setSiteType(x,y,z,site);
    isGPUMemorySynched = false;
}

void CudaIntLattice::addParticle(lattice_size_t x, lattice_size_t y, lattice_size_t z, particle_t particle) 
{
    IntLattice::addParticle(x,y,z,particle);
	isGPUMemorySynched = false;
}

void CudaIntLattice::removeParticles(lattice_size_t x,lattice_size_t y,lattice_size_t z) 
{
    IntLattice::removeParticles(x,y,z);
    isGPUMemorySynched = false;
}

void CudaIntLattice::setSiteType(lattice_size_t index, site_t site) 
{
    IntLattice::setSiteType(index,site);
    isGPUMemorySynched = false;
}

void CudaIntLattice::addParticle(lattice_size_t index, particle_t particle) 
{
    IntLattice::addParticle(index,particle);
	isGPUMemorySynched = false;
}

void CudaIntLattice::removeParticles(lattice_size_t index) 
{
    IntLattice::removeParticles(index);
    isGPUMemorySynched = false;
}

void CudaIntLattice::removeAllParticles()
{
    IntLattice::removeAllParticles();
	isGPUMemorySynched = false;
}

void CudaIntLattice::setFromRowMajorByteData(void * buffer, size_t bufferSize)
{
    IntLattice::setFromRowMajorByteData(buffer, bufferSize);
    isGPUMemorySynched = false;
}

}
}
