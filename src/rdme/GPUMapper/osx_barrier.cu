
#include <hip/hip_runtime.h>
/*
 * University of Illinois Open Source License
 * Copyright 2012-2018 Luthey-Schulten Group,
 * All rights reserved.
 * 
 * Developed by: Luthey-Schulten Group
 * 			     University of Illinois at Urbana-Champaign
 * 			     http://www.scs.uiuc.edu/~schulten
 * 
 * Permission is hereby granted, free of charge, to any person obtaining a copy of
 * this software and associated documentation files (the Software), to deal with 
 * the Software without restriction, including without limitation the rights to 
 * use, copy, modify, merge, publish, distribute, sublicense, and/or sell copies 
 * of the Software, and to permit persons to whom the Software is furnished to 
 * do so, subject to the following conditions:
 * 
 * - Redistributions of source code must retain the above copyright notice, 
 * this list of conditions and the following disclaimers.
 * 
 * - Redistributions in binary form must reproduce the above copyright notice, 
 * this list of conditions and the following disclaimers in the documentation 
 * and/or other materials provided with the distribution.
 * 
 * - Neither the names of the Luthey-Schulten Group, University of Illinois at
 * Urbana-Champaign, nor the names of its contributors may be used to endorse or
 * promote products derived from this Software without specific prior written
 * permission.
 * 
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR 
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY, 
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT.  IN NO EVENT SHALL 
 * THE CONTRIBUTORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR 
 * OTHER LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, 
 * ARISING FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR 
 * OTHER DEALINGS WITH THE SOFTWARE.
 *
 * Author(s): Mike Hallock
 */

#if defined(MACOSX)

#include <pthread.h>
#include "config.h"
#include "osx_barrier.h"

int pthread_barrier_init(pthread_barrier_t *b, void *attrs, int count)
{
	b->participants=count;
	b->current=0;
	pthread_mutex_init(&b->mutex,NULL);
	pthread_cond_init(&b->condition,NULL);
	return 0;
}

int pthread_barrier_wait(pthread_barrier_t *b)
{
	pthread_mutex_lock(&b->mutex);
	int rv=0;
	b->current++;
	if(b->current == b->participants)
	{
		b->current=0;
		pthread_cond_broadcast(&b->condition);
		rv=PTHREAD_BARRIER_SERIAL_THREAD;
	}	
	else
	{
		pthread_cond_wait(&b->condition, &b->mutex);
	}

	pthread_mutex_unlock(&b->mutex);
	return rv;
}

#endif
